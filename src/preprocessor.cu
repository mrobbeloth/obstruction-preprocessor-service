#include <opencv2/opencv.hpp>
#include <iostream>
#include <SuperString.hh>
#include <string>
#include <thread>

#include "utility.h"
#include "threadpool.hpp"
#include "preprocessorKernels.h"

using namespace std;
using namespace cv;
using namespace astp;

void threadFunc(const int &i) {
    // Allocate memory on device
    const int* in = &i;
    int* out = nullptr;
    auto err = hipMalloc((void **)&out, sizeof(int));

    // Copy to device memory
    err = hipMemcpy(out, in, sizeof(int), hipMemcpyHostToDevice);

    // Call kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = threadsPerBlock / threadsPerBlock;
    cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads" << endl;
    multiplyByTen<<<blocksPerGrid, threadsPerBlock>>>(in, out);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch multiplyByTen kernel (%s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    } else {
        cout << "ThreadPool " << i << endl;
    }
}

int main() {
    ThreadPool tp; 
    for (int i = 0; i < 100; i++) {
        tp.push([i]() {
            threadFunc(i);
        });
    }

    tp.wait();
    return 0;
}























/*
    ThreadPool pool(500);
    std::vector<std::future<thread::id>> results;

    auto now = system_clock::now();
    auto now_ms = time_point_cast<milliseconds>(now);

    auto value = now_ms.time_since_epoch();

    int processes = 10;
    int finish_flags = 0;
    while (finish_flags < processes) {
        for (int i = 0; i < processes; ++i) {
            auto future = pool.enqueue([i] {
                std::this_thread::sleep_for(std::chrono::milliseconds(100));
                cout << i << endl;
                return this_thread::get_id();
            });
            finish_flags++;
            results.emplace_back(std::move(future));
        }
    }

    auto new_now = system_clock::now();
    auto new_now_ms = time_point_cast<milliseconds>(new_now);

    auto new_value = new_now_ms.time_since_epoch();
    long time_taken = (new_value - value).count();

    cout << "Time taken: " << time_taken << endl;

    return 0;
    */